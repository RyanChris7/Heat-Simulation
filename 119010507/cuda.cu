#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <chrono>

#ifdef GUI
#include <GL/glut.h>
#include <GL/gl.h>
#include <GL/glu.h>
#endif

#include "./headers/physics.h"


int block_size = 512; // cuda thread block size
__device__ int size; // problem size


__global__ void initialize(float *data) {
    // intialize the temperature distribution
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int len = size * size;
    if (i < len) {
        data[i] = wall_temp;
    }
}


__global__ void generate_fire_area(bool *fire_area){
    // TODO: generate the fire area (in parallelized way)
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int len = size * size;
    if (i < len) {
        fire_area[i] = 0;
    }

    float fire1_r2 = fire_size * fire_size;
    for (int i = 0; i < size; i++){
        for (int j = 0; j < size; j++){
            int a = i - size / 2;
            int b = j - size / 2;
            int r2 = 0.5 * a * a + 0.8 * b * b - 0.5 * a * b;
            if (r2 < fire1_r2) fire_area[i * size + j] = 1;
        }
    }

    float fire2_r2 = (fire_size / 2) * (fire_size / 2);
    for (int i = 0; i < size; i++){
        for (int j = 0; j < size; j++){
            int a = i - 1 * size / 3;
            int b = j - 1 * size / 3;
            int r2 = a * a + b * b;
            if (r2 < fire2_r2) fire_area[i * size + j] = 1;
        }
    }
}


__global__ void update(float *data, float *new_data) {
    // TODO: update temperature for each point  (in parallelized way)
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < size - 1){
        for (int j = 1; j < size - 1; j++){
            int idx = i * size + j;
            float up = data[idx - size];
            float down = data[idx + size];
            float left = data[idx - 1];
            float right = data[idx + 1];
            float new_val = (up + down + left + right) / 4;
            new_data[idx] = new_val;
        }
    }
}


__global__ void maintain_wall(float *data) {
    // TODO: maintain the temperature of the wall (sequential is enough)
    for (int i = 0; i< size; i++){
        data[i] = wall_temp;
    }
    for (int j = 0; j< size; j++){
        data[j * size] = wall_temp;
    }
    for (int k = 0; k< size; k++){
        data[(k*size) + (size -1)] = wall_temp;
    }
}


__global__ void maintain_fire(float *data, bool *fire_area) {
    // TODO: maintain the temperature of the fire (in parallelized way)
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int len = size * size;
    if (i < len){
        if (fire_area[i]) data[i] = fire_temp;
    }
}


#ifdef GUI
__global__ void data2pixels(float *data, GLubyte* pixels){
    // TODO: convert rawdata (large, size^2) to pixels (small, resolution^2) for faster rendering speed (in parallelized way)
}


void plot(GLubyte* pixels){
    // visualize temprature distribution
    #ifdef GUI
    glClear(GL_COLOR_BUFFER_BIT);
    glDrawPixels(resolution, resolution, GL_RGB, GL_UNSIGNED_BYTE, pixels);
    glutSwapBuffers();
    #endif
}
#endif


void master() {
    float *data_odd;
    float *data_even;
    bool *fire_area;

    hipMalloc(&data_odd, size * size * sizeof(float));
    hipMalloc(&data_even, size * size * sizeof(float));
    hipMalloc(&fire_area, size * size * sizeof(bool));

    #ifdef GUI
    GLubyte *pixels;
    GLubyte *host_pixels;
    host_pixels = new GLubyte[resolution * resolution * 3];
    hipMalloc(&pixels, resolution * resolution * 3 * sizeof(GLubyte));
    #endif

    int n_block_size = size * size / block_size + 1;
    int n_block_resolution = resolution * resolution / block_size + 1;

    initialize<<<n_block_size, block_size>>>(data_odd);
    generate_fire_area<<<n_block_size, block_size>>>(fire_area);
    
    int count = 1;
    double total_time = 0;

    while (true){
        std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();

        // TODO: modify the following lines to fit your need.
        if (count % 2 == 1) {
            update<<<n_block_size, block_size>>>(data_odd, data_even);
            maintain_fire<<<n_block_size, block_size>>>(data_even, fire_area);
            maintain_wall<<<1, 1>>>(data_even);
        } else {
            update<<<n_block_size, block_size>>>(data_even, data_odd);
            maintain_fire<<<n_block_size, block_size>>>(data_odd, fire_area);
            maintain_wall<<<1, 1>>>(data_odd);
        }

        std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();
        double this_time = std::chrono::duration<double>(t2 - t1).count();
        total_time += this_time;
        printf("Iteration %d, elapsed time: %.6f\n", count, this_time);
        count++;
        
        #ifdef GUI
        if (count % 2 == 1) {
            data2pixels<<<n_block_resolution, block_size>>>(data_even, pixels);
        } else {
            data2pixels<<<n_block_resolution, block_size>>>(data_odd, pixels);
        }
        hipMemcpy(host_pixels, pixels, resolution * resolution * 3 * sizeof(GLubyte), hipMemcpyDeviceToHost);
        plot(host_pixels);
        #endif

    }

    printf("Converge after %d iterations, elapsed time: %.6f, average computation time: %.6f\n", count-1, total_time, (double) total_time / (count-1));


    hipFree(data_odd);
    hipFree(data_even);
    hipFree(fire_area);

    #ifdef GUI
    hipFree(pixels);
    delete[] host_pixels;
    #endif
    
}


int main(int argc, char *argv[]){
    
    size = atoi(argv[1]);

    #ifdef GUI
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_RGB | GLUT_SINGLE);
    glutInitWindowPosition(0, 0);
    glutInitWindowSize(resolution, resolution);
    glutCreateWindow("Heat Distribution Simulation Sequential Implementation");
    gluOrtho2D(0, resolution, 0, resolution);
    #endif

    master();

    printf("Student ID: 119010001\n"); // replace it with your student id
    printf("Name: Your Name\n"); // replace it with your name
    printf("Assignment 4: Heat Distribution CUDA Implementation\n");

    return 0;

}


